#include "hip/hip_runtime.h"
#include "cuBase.h"

__device__ float d_nonLinearity(float val, int NONLIN){
	if(NONLIN == NL_RELU){
		if(val < 0.0) return 0.0;
		else return val;
	}else if(NONLIN == NL_LRELU){
        if(val < 0.0) return 0.1f * val;
        else return val;
    }else if(NONLIN == NL_TANH){
		return tanh(val * 2.0 / 3.0) * 1.7159;
	}
	else{
		return val;
	}
}

__device__ float d_dnonLinearity(float val,int NONLIN){
	if(NONLIN == NL_RELU){
		if(val > 0.0) return 1.0;
		else return 0.0;
	}else if (NONLIN == NL_LRELU){
        if(val > 0.0) return 1.0;
        else return 0.1;
    }
	else if(NONLIN == NL_TANH){
		float res = 1.7159;
		float temp = val * val / 1.7159;
		res = (res - temp) * 2.0 / 3.0;
		return res;
	}else {
		return val;
	}
}

/* given each input and output spike train of spike times, 
 * compute the accumulative synaptic effect
 * input: input spikes: endTime * inputDim
 * output: output spikes: endTime * outputDim
 */
__device__ float d_Spiking_accumulate_effect(
    int* output_time,
    int* input_time,
    int n_ospikes,
    int n_ispikes,
    int o_idx,
    int i_idx,
    int outputDim,
    int inputDim,
    int endTime,
    int T_REFRAC,
    float TAU_M,
    float TAU_S)
{
    float acc_response = 0.0f;
    int t_post_last = 1;
    for(int i = 0; i < n_ospikes; ++i){
        int t_post = output_time[o_idx * endTime + i];
        float sum = 0.0f;
        
        int ub = t_post;
        int lb = max(1, int(t_post - 4*TAU_M));
        for(int j = 0; j < n_ispikes; ++j){
            int t_pre = input_time[i_idx * endTime + j];
            if(t_pre < lb)    continue;
            if(t_pre >= ub)    break;

            int pre_time = t_pre + T_REFRAC;
            if(pre_time > t_post)   continue;
            int s = t_post - t_post_last;
            int t = t_post - pre_time;
            float factor = __expf(-1*max(t - s, 0)/TAU_S)/(1 - TAU_S/TAU_M);
            sum += factor * (__expf(-1*min(s, t)/TAU_M) - __expf(-1*min(s, t)/TAU_S));
        }
        t_post_last = t_post + T_REFRAC;
        acc_response += sum;
    }
    if(n_ospikes == 0 && n_ispikes != 0)
        acc_response = 0.1;
    return acc_response;
}


__global__ void g_dnonLinearity(float* delta, float*acti, int len, int NONLIN)
{
	int skip = gridDim.x * blockDim.x;
	for(int i = 0; i < len; i += skip)
	{
		int id = blockDim.x * blockIdx.x + threadIdx.x + i;
		if(id < len)
		{	
			delta[id] *= d_dnonLinearity(acti[id], NONLIN);
		}
	}
}

__global__ void g_nonLinearity(float* inputs, int len, int NONLIN)
{
	for(int i = 0; i < len; i += gridDim.x * blockDim.x)
	{
		int id = blockDim.x * blockIdx.x + threadIdx.x + i;
		if(id < len)
		{	
			inputs[id] = d_nonLinearity(inputs[id], NONLIN);
		}
	}
}

__device__ void swap(float& val1, float& val2){
	float tmp = val1;
	val1 = val2;
	val2 = tmp;
}


__global__ void g_vecAdd(float**_v_w, float** _wgrad,float** _w,
	float** _v_b, float** _bgrad, float** _b, 
	int lenw, int lenb,
	float momentum, float lratew, float lrateb)
{
	float* v_w   = _v_w[blockIdx.x];
	float* wgrad = _wgrad[blockIdx.x];
	float* w     = _w[blockIdx.x];
	float* v_b   = _v_b[blockIdx.x];
	float* bgrad = _bgrad[blockIdx.x];
	float* b     = _b[blockIdx.x];

	int idx = threadIdx.x;
	for(int i = 0; i < lenw; i += blockDim.x)
	{
		int id = i + idx;
		if(id < lenw)
		{
			v_w[id] = v_w[id] * momentum + wgrad[id] * lratew;
			w[id] -= v_w[id];
		}
	}
	for(int i = 0; i < lenb; i += blockDim.x)
	{
		int id = i + idx;
		if(id < lenb)
		{
			v_b[id] = v_b[id] * momentum + bgrad[id] * lrateb;
			b[id] -= v_b[id];
		}
	}
}

__global__ void g_vecAdd(float*v_w, float*wgrad,float* w,
	float* v_b, float* bgrad, float* b, 
	int lenw, int lenb,
	float momentum, float lratew, float lrateb)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	for(int i = 0; i < lenw; i += blockDim.x * gridDim.x)
	{
		int id = i + idx;
		if(id < lenw)
		{
			v_w[id] = v_w[id] * momentum + wgrad[id] * lratew;
			w[id] -= v_w[id];
		}
	}
	for(int i = 0; i < lenb; i += blockDim.x * gridDim.x)
	{
		int id = i + idx;
		if(id < lenb)
		{
			v_b[id] = v_b[id] * momentum + bgrad[id] * lrateb;
			b[id] -= v_b[id];
		}
	}
}

/*
 * block  = dim3(outputAmount)
 * thread = dim3(min(256, w[0]->getLen()))
 */
__global__ void g_sgd_vecAdd(float** momentum_w, float** _wgrad, float** _w, int lenw, float momentum, float lr)
{
    int ok = blockIdx.x;
    float* v_w   = momentum_w[ok];
    float* w     = _w[ok];
    float* wgrad = _wgrad[ok];
    int idx = threadIdx.x;
    for(int i = 0; i < lenw; i += blockDim.x)
    {
        int id = i + idx;
        if(id < lenw)
        {
            v_w[id] = v_w[id] * momentum + wgrad[id] * lr;
            w[id]  -= v_w[id];
        }
    }
}


/*
 * block  = dim3(outputAmount)
 * thread = dim3(min(256, w[0]->getLen()))
 */
__global__ void g_adam_vecAdd(float** g1_ws, float** g2_ws, float* b1_t, float* b2_t, float** _wgrad, float** _w, int lenw, float lr)
{
    int ok = blockIdx.x;
    float* g1_w  = g1_ws[ok];
    float* g2_w  = g2_ws[ok];
    float* w     = _w[ok];
    float* wgrad = _wgrad[ok];
    int idx = threadIdx.x;
    float b1t = b1_t[ok];
    float b2t = b2_t[ok];
    const float b1 = 0.9f;
    const float b2 = 0.999f;
    const float eps = 1.e-8f;
    __syncthreads();

    for(int i = 0; i < lenw; i += blockDim.x)
    {
        int id = i + idx;
        if(id < lenw)
        {
            float weight_grad = wgrad[id];
            float g1 = b1 * g1_w[id] + (1 - b1) * weight_grad;
            float g2 = b2 * g2_w[id] + (1 - b2) * weight_grad * weight_grad;
            w[id]  -= lr * (g1/(1.f - b1t)) / ((float)sqrtf(g2/(1. - b2t)) + eps);
            g1_w[id] = g1;
            g2_w[id] = g2;
        }
    }
    if(threadIdx.x == 0){
        b1_t[ok] *= b1;
        b2_t[ok] *= b2;
    }
}

/* Use this function when outputAmount = 1
 * block  = dim3(min((w->getLen() + 255)/256, 5120))
 * thread = dim3(256)
 */
__global__ void g_sgd_vecAdd(float* v_w, float* wgrad, float* w, int lenw, float momentum, float lr)
{
   int idx = blockDim.x * blockIdx.x + threadIdx.x;
   for(int i = 0; i < lenw; i += blockDim.x * gridDim.x)
   {
       int id = i + idx;
       if(id < lenw)
       {
           v_w[id] = v_w[id] * momentum + wgrad[id] * lr;
           w[id]  -= v_w[id];
       }
   }
}

/* Use this function when outputAmount = 1
 * block  = dim3(min((w->getLen() + 255)/256, 5120))
 * thread = dim3(256)
 */
__global__ void g_adam_vecAdd(float* g1_w, float* g2_w, float b1t, float b2t, float* wgrad, float* w, int lenw, float lr)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    const float b1 = 0.9f;
    const float b2 = 0.999f;
    const float eps = 1.e-8f;
    
    for(int i = 0; i < lenw; i += blockDim.x * gridDim.x)
    {
        int id = i + idx;
        if(id < lenw)
        {
            float weight_grad = wgrad[id];
            float g1 = b1 * g1_w[id] + (1 - b1) * weight_grad;
            float g2 = b2 * g2_w[id] + (1 - b2) * weight_grad * weight_grad;
            w[id]  -= lr * (g1/(1.f - b1t)) / ((float)sqrtf(g2/(1. - b2t)) + eps);
            g1_w[id] = g1;
            g2_w[id] = g2;
        }
    }
}

/* Use this function when outputAmount = 1
 * block  = dim3(min((w->getLen() + 255)/256, 5120))
 * thread = dim3(256)
 */
__global__ void g_adam_vecAdd_reservoir(float* g1_w, float* g2_w, float b1t, float b2t, float* wgrad, float* w_l, int lenw, float lr)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    const float b1 = 0.9f;
    const float b2 = 0.999f;
    const float eps = 1.e-8f;
    
    for(int i = 0; i < lenw; i += blockDim.x * gridDim.x)
    {
        int id = i + idx;
        if(id < lenw)
        {
			if(w_l[id]!=0){
				float weight_grad = wgrad[id];
				float g1 = b1 * g1_w[id] + (1 - b1) * weight_grad;
				float g2 = b2 * g2_w[id] + (1 - b2) * weight_grad * weight_grad;
				w_l[id]  -= lr * (g1/(1.f - b1t)) / ((float)sqrtf(g2/(1. - b2t)) + eps);
				g1_w[id] = g1;
				g2_w[id] = g2;
			}
        }
    }
}


__global__ void g_getCost_3(float* cost,
	float** weight,
	float lambda, int wlen)
{
	extern __shared__ float _sum[];
	_sum[threadIdx.x] = 0;
	__syncthreads();
	float* w = weight[blockIdx.x];

	for(int i = 0; i < wlen; i += blockDim.x)
	{
		int id = i + threadIdx.x;
		if(id < wlen)
		{
			_sum[threadIdx.x] += w[id] * w[id];
		}
	}

	int len = blockDim.x;
	while(len != 1)
	{
		__syncthreads();
		int skip = (len + 1) >> 1;
		if(threadIdx.x < skip && (threadIdx.x + skip) < len)
		{
			_sum[threadIdx.x] += _sum[threadIdx.x + skip];
		}
		len = skip;
	}

	if(threadIdx.x == 0)
	{
		atomicAdd(cost, _sum[0] * lambda * 0.5);
	}
}


/*
*/
__global__ void g_getBgrad(float* softMaxDelta, float* bgrad, float* dropb, int batch)
{
	extern __shared__ float _sum[];
	_sum[threadIdx.x] = softMaxDelta[threadIdx.x * gridDim.x + blockIdx.x];

	int len = blockDim.x;
	while(len != 1)
	{
		__syncthreads();
		int skip = (len + 1) >> 1;
		if(threadIdx.x < skip && (threadIdx.x + skip) < len)
		{
			_sum[threadIdx.x] += _sum[threadIdx.x + skip];
		}
		len = skip;
	}
	if(threadIdx.x == 0)
	{
		bgrad[blockIdx.x] = _sum[0] / batch;
		bgrad[blockIdx.x] *= dropb[blockIdx.x];
	}
}


/*
dim3(curDelta->cols), dim3(curDelta->rows), 
sizeof(float) * curDelta->rows
*/
__global__ void g_getBgrad(float* softMaxDelta, float* bgrad, int batch)
{
	extern __shared__ float _sum[];
	_sum[threadIdx.x] = softMaxDelta[threadIdx.x * gridDim.x + blockIdx.x];

	int len = blockDim.x;
	while(len != 1)
	{
		__syncthreads();
		int skip = (len + 1) >> 1;
		if(threadIdx.x < skip && (threadIdx.x + skip) < len)
		{
			_sum[threadIdx.x] += _sum[threadIdx.x + skip];
		}
		len = skip;
	}
	__syncthreads();
	if(threadIdx.x == 0)
	{
		bgrad[blockIdx.x] = _sum[0] / batch;
	}
}

/*
* function: getcost
*/
__global__ void g_getCost_1(float* softMaxP,
	float* groundTruth, float* cost, int*y, int rows, int cols, int batch)
{
	extern __shared__ float _sum[];
	int len = rows * cols;
	for(int i = 0; i < len; i += blockDim.x)
	{
		int id = i + threadIdx.x;
		if(id < len)
		{
			groundTruth[id] = 0;
		}
	}
	__syncthreads();
	for(int i = 0; i < rows; i += blockDim.x)
	{
		int id = i + threadIdx.x;
		if(id < rows)
		{
			int yy = y[id];
			groundTruth[id * cols + yy] = 1;
		}
	}
	_sum[threadIdx.x] = 0;
	__syncthreads();
	for(int i = 0; i < len; i += blockDim.x)
	{
		int id = i + threadIdx.x;
		if(id < len)
		{
			_sum[threadIdx.x] += __logf(softMaxP[id] + 1.0e-10) * groundTruth[id];
		}
	}
	len = blockDim.x;
	while(len != 1)
	{
		__syncthreads();
		int skip = (len + 1) >> 1;
		if(threadIdx.x < skip && (threadIdx.x + skip) < len)
		{
			_sum[threadIdx.x] += _sum[threadIdx.x + skip];
		}
		len = skip;
	}
	__syncthreads();
	if(threadIdx.x == 0)
	{
		cost[0] = -_sum[0] / batch;
	}
}


__global__ void g_getCost_2(float* cost,
	float* weight,
	float lambda, int len)
{
	extern __shared__ float _sum[];
	_sum[threadIdx.x] = 0;
	__syncthreads();
	for(int i = 0; i < len; i += blockDim.x)
	{
		int id = i + threadIdx.x;
		if(id < len)
		{
			_sum[threadIdx.x] += 0.5 * weight[id] * weight[id];
		}
	}
	len = blockDim.x;
	while(len != 1)
	{
		__syncthreads();
		int skip = (len + 1) >> 1;
		if(threadIdx.x < skip && (threadIdx.x + skip) < len)
		{
			_sum[threadIdx.x] += _sum[threadIdx.x + skip];
		}
		len = skip;
	}
	__syncthreads();
	if(threadIdx.x == 0)
	{
		cost[0] += _sum[0] * lambda;
	}
}


/*
* function: cuMatrix(batch, channel * size, 1) to cuMatrix(batch, size, channel)
* blocks  : dim3(batch)
* threads : dim3(min(512, size * channels))
*/
__global__ void g_preDeltaFormat(float* cuPoolFlDelta, 
	float* cuPoolDelta, int batch, int size, int channels){
	int b = blockIdx.x;
	int len = size * channels;
	for(int i = 0; i < len; i += blockDim.x){
		int id = i + threadIdx.x;
		if(id < len){
			int s = id / channels;
			int c = id % channels;
			cuPoolDelta[c * batch * size + b * size + s] = cuPoolFlDelta[b * size * channels + size * c + s];
		}
	}
}


/*
* function: cuMatrix(batch, size, channel) to cuMatrix(batch, channel * size, 1)
* blocks  : dim3(batch)
* threads : dim3(min(512, cuPool[poolidx]->cols))
*/
__global__ void g_convert(float* cuPool, float*cuPoolToFlActi, int batch, int size, int channel){
	int b   = blockIdx.x;
	int len = size * channel;
	for(int i = 0; i < len; i+=blockDim.x){
		int id = i + threadIdx.x;
		if(id < len){
			int s = id / channel;
			int c = id % channel;
			cuPoolToFlActi[b * size * channel + size * c + s] = cuPool[c * batch * size + b * size + s];
		}
	}
}

/*
* 
* function: cuMatrix<int>*(batch, inputDim2*endTime, amount) 
*           to cuMatrix<int>*(batch, amount*inputDim2*endTime, 1)
* Notice that the inputDim is the one dim of the image if amount > 1 (CNN case for img)
*
*   inputSize = amount * inputDim*inputDim
*   inputCols = endTime * inputDim*inputDim
*    channels = amount
*
* blocks  : dim3(batch, endTime)
* threads : dim3(min(1024, inputSize))
*/
__global__ void g_convert_spiketimes(int* inputs_time, int endTime, int inputSize, int inputCols, int channels, int batch, int* inputs_tf){
    int b = blockIdx.x;
    int t = blockIdx.y;
    for(int i = 0; i < inputSize; i += blockDim.x){
        int i_idx = i + threadIdx.x;
        if(i_idx < inputSize){
            int s = i_idx / channels;
            int c = i_idx % channels;
            int index = c * batch * inputCols + b * inputCols + s*endTime + t;
            inputs_tf[b * inputCols * channels + c*inputCols + s*endTime+ t] = inputs_time[index];
        }
    }
}

/*
* 
* function: cuMatrix<int>*(batch, inputDim2, amount) 
*           to cuMatrix<int>*(batch, amount*inputDim2, 1)
* Notice that the inputDim is the one dim of the image if amount > 1 (CNN case for img)
*
*   inputSize = amount * inputDim*inputDim
*   inputDim2 = inputDim*inputDim
*    channels = amount
*
* blocks  : dim3(batch)
* threads : dim3(min(1024, inputSize))
*/
__global__ void g_convert_firecounts(int* counts, int area, int inputSize, int inputDim2, int channels, int batch, int* counts_f){
    int b = blockIdx.y;
    for(int i = 0; i < inputSize; i += blockDim.x){
        int i_idx = i + threadIdx.x;
        if(i_idx < inputSize){
            int s = i_idx / channels;
            int c = i_idx % channels;
            counts_f[b*inputDim2*channels + c*inputDim2 + s] = counts[c*area + b*inputDim2 + s];
        }
    } 
}
/*
* 
* function: cuMatrix<bool>*(batch, endTime*inputDim*inputDim, amount) 
*           to cuMatrix<bool>*(inputSize, endTime*batch, 1)
* Notice that the inputDim is the one dim of the image if amount > 1 (CNN case for img)
*
*   inputSize = amount * inputDim*inputDim
*   inputCols = endTime * inputDim*inputDim
*    channels = amount
*
* blocks  : dim3(batch, endTime)
* threads : dim3(min(1024, inputSize))
*/
__global__ void g_cast_bool_2_float(bool* inputs, int endTime, int inputSize, int inputCols, int channels, int batch, float* inputs_f){
	int b   = blockIdx.x;
    int t   = blockIdx.y;
    int inputDim2 = inputCols / endTime;
	for(int i = 0; i < inputSize; i += blockDim.x){
		int i_idx = i + threadIdx.x;
		if(i_idx < inputSize){
            int s = i_idx / channels; // the index for inputDim2, within the same channel
            int c = i_idx % channels;
            int index = c * batch * inputCols + b * inputCols + t * inputDim2 + s;
            inputs_f[(c * inputDim2 + s) * endTime * batch + t * batch + b] = inputs[index];
		}
	}
}


/*
* 
* function: cuMatrix<float>*(outputSize, endTime*batch) to cuMatrix<float>*(batch, outputSize*endTime)
* blocks  : dim3(batch, outputSize)
* threads : dim3(min(1024, endTime))
*/
__global__ void g_transform_2_batch(float* inputs_rt, int endTime, int outputSize, int batch, float* inputs_r){
	int b     = blockIdx.x;
    int o_idx = blockIdx.y;
    int size2 = outputSize * endTime;
    float* input_r = inputs_r + b * size2;
	for(int t = 0; t < endTime; t += blockDim.x){
		int time = t + threadIdx.x;
		if(time < endTime){
            input_r[o_idx * endTime + time] = inputs_rt[o_idx * endTime * batch + time * batch + b];
		}
	}
}

/*
* function: transform the binary response matrix (batch, outputSize * endTime, outputAmount) 
* to spike times matrix (batch, outputSize*"num of spikes", outputAmount),directly store the spike times. 
* blocks  : dim3(batch, outputAmount)
* threads : dim3(min(1024, outputSize))
*/
__global__ void g_response_2_spiketime(bool* outputs, int* outputs_time, int outputArea, int outputSize, int endTime)
{
    int batchId = blockIdx.x;
    int ok = blockIdx.y;
    bool* output = outputs + ok * outputArea + batchId * endTime * outputSize;
    int* output_time = outputs_time + ok * outputArea + batchId * endTime * outputSize;

    for(int i = 0; i < outputSize; i += blockDim.x)
    {
        int o_idx = i + threadIdx.x;
        if(o_idx < outputSize)
        {
            int col_idx = 0;
            for(int time = 0; time < endTime; ++time)
            {
                if(output[o_idx + time * outputSize])
                {
                    output_time[o_idx * endTime + col_idx] = time;
                    col_idx++;
                }
            }
        }
    }
}

/*
* function: divide the curDelta(batch, outputSize, outputAmount) by vth
* blocks  : dim3(batch, outputAmount)
* threads : dim3(min(1024, outputSize))
*/
__global__ void g_divide_by_threshold(float * _delta, int area, int outputSize, float threshold)
{
    int batchId = blockIdx.x;
    int ok = blockIdx.y;
    float * delta = _delta + ok * area + batchId * outputSize;
    for (int tidx = 0; tidx < outputSize; tidx += blockDim.x) {
        int o_idx = tidx + threadIdx.x;
        if (o_idx < outputSize) {
            delta[o_idx] /= threshold;
        }
    }
}


/*
* blocks : cuSoftMaxP->rows
* threads: cuSoftMaxP->cols
* shared : sizeof(float) * cuSoftMaxP->cols * 2
*/
__global__ void g_getSoftMaxP(float* softMaxP, float* b, int cols)
{
	int bid = blockIdx.x;
	extern __shared__ float _share[];
	float * _max = _share;
	float * _sum = _share + blockDim.x;
	float* sp = softMaxP + bid * cols;
	_sum[threadIdx.x] = 0.0;
	_max[threadIdx.x] = -100000000.0;
	for(int tid = 0; tid < cols; tid += blockDim.x){
		int id = tid + threadIdx.x;
		if(id < cols){
			sp[id] += b[id];
			_max[threadIdx.x] = max(_max[threadIdx.x], sp[id]);
		}
	}
	__syncthreads();
	int len = blockDim.x;
	while(len != 1)
	{
		__syncthreads();
		int skip = (len + 1) >> 1;
		if(threadIdx.x < (len >> 1))
		{
			if(_max[threadIdx.x] < _max[threadIdx.x + skip])
			{
				_max[threadIdx.x] = _max[threadIdx.x + skip];
			}
		}
		len = (len + 1) >> 1;
	}
	__syncthreads();
	for(int tid = 0; tid < cols; tid += blockDim.x){
		int id = tid + threadIdx.x;
		if(id < cols){
			sp[id] -= _max[0];
			sp[id] = __expf(sp[id]);
			_sum[threadIdx.x] += sp[id];
		}
	}
	__syncthreads();
	len = blockDim.x;
	while(len != 1)
	{
		__syncthreads();
		int skip = (len + 1) >> 1;
		if(threadIdx.x < (len >> 1))
		{
			_sum[threadIdx.x] += _sum[threadIdx.x + skip];
		}
		len = (len + 1) >> 1;
	}
	__syncthreads();
	for(int tid = 0; tid < cols; tid += blockDim.x){
		int id = tid + threadIdx.x;
		if(id < cols){
			sp[id] /= _sum[0];
		}
	}
}

__global__ void g_getSoftMaxDelta(float* softMaxDelta, float* softMaxP, float* groundTruth, int len)
{
	for(int i = 0; i < len; i += blockDim.x)
	{
		int id = i + threadIdx.x;
		if(id < len)
		{
			softMaxDelta[id] = softMaxP[id] - groundTruth[id];
		}
	}
}

__global__ void g_getSmrWgrad(float* wgrad, float* weight, float lambda, int len, int batch)
{
	for(int i = 0; i < len; i += blockDim.x)
	{
		int id = i + threadIdx.x;
		if(id < len)
		{
			wgrad[id] = lambda * weight[id] + wgrad[id] / batch;
		}
	}
}


/* Use this function when outputAmount = 1
 * block  = dim3(1)
 * thread = dim3(outputSize)
 */
__global__ void g_adam_self_vecAdd(float* g1_w, float* g2_w, float b1t, float b2t, float* wgrad_s, float* w_s, int lenw, float lr)
{
    int idx = threadIdx.x;
    const float b1 = 0.9f;
    const float b2 = 0.999f;
    const float eps = 1.e-8f;
    
    for(int i = 0; i < lenw; i += blockDim.x)
    {
        int id = i + idx;
        if(id < lenw)
        {
            float weight_grad_s = wgrad_s[id];
            float g1 = b1 * g1_w[id] + (1 - b1) * weight_grad_s;
            float g2 = b2 * g2_w[id] + (1 - b2) * weight_grad_s * weight_grad_s;
            w_s[id]  -= lr * (g1/(1.f - b1t)) / ((float)sqrtf(g2/(1. - b2t)) + eps);
            g1_w[id] = g1;
            g2_w[id] = g2;
        }
    }
}

/* given each input and output spike train of spike times, 
 * compute the accumulative synaptic effect
 * input: input spikes: endTime * inputDim
 * output: output spikes: endTime * outputDim
 */
__device__ float d_Spiking_accumulate_effect_step(
    int* output_time,
    int* input_time,
    int n_ospikes,
    int n_ispikes,
    int o_idx,
    int i_idx,
    int outputDim,
    int inputDim,
    int endTime,
    int T_REFRAC,
    float TAU_M,
    float TAU_S)
{
    float eij = 0.0f;
	float p=0;
	float q=0;
	int index_in=0;
	int index_out=0;
	int t_ref=0;
	if(n_ispikes == 0)
		return 0;
    if(n_ospikes == 0)
		return 0;
	for(int t = 0; t < endTime; t++){
		p-=p/TAU_S;
		if(index_in<n_ispikes && input_time[i_idx*endTime+index_in]==t-1){
			p+=1;
			index_in++;
		}
		q-=q/TAU_M;
		q+=p/TAU_S;
		if(t_ref!=0){
			q=0;
			t_ref--;
		}
		if(index_out<n_ospikes && output_time[o_idx*endTime+index_out]==t){
			eij+=q;
			index_out++;
			t_ref=T_REFRAC;
			q=0;
		}
		if(index_out>=n_ospikes){
			break;
		}
	}
	//printf("a: %f\n", record_effect);
	//printf("b: %f\n", record_v_effect);
	//printf("eij: %f\n", acc_response);
    return eij;
}


