#include "hip/hip_runtime.h"
#include "dataPretreatment.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "../common/cuMatrix.h"
#include "../common/cuMatrixVector.h"

__global__ void g_getAver(float ** input1,
	float** input2, 
	float* aver,
	int num_of_input1, 
	int num_of_input2,
	int imgSize)
{
	for(int j = 0; j < num_of_input1; j++)
	{
		for(int i = 0; i < imgSize; i += blockDim.x)
		{
			int idx = threadIdx.x + i;
			if(idx < imgSize)
			{
				aver[idx] += input1[j][idx];
			}
		}
	}

	__syncthreads();

	for(int j = 0; j < num_of_input2; j++)
	{
		for(int i = 0; i < imgSize; i += blockDim.x)
		{
			int idx = threadIdx.x + i;
			if(idx < imgSize)
			{
				aver[idx] += input2[j][idx];
			}
		}
	}

	__syncthreads();

	for(int i = 0; i < imgSize; i += blockDim.x)
	{
		int idx = i + threadIdx.x;
		if(idx < imgSize)
		{
			aver[idx] /= (num_of_input1 + num_of_input2);
		}
	}

	__syncthreads();


	for(int j = 0; j < num_of_input1; j++)
	{
		for(int i = 0; i < imgSize; i += blockDim.x)
		{
			int idx = threadIdx.x + i;
			if(idx < imgSize)
			{
				input1[j][idx] -= aver[idx];
			}
		}
	}

	__syncthreads();

	for(int j = 0; j < num_of_input2; j++)
	{
		for(int i = 0; i < imgSize; i += blockDim.x)
		{
			int idx = threadIdx.x + i;
			if(idx < imgSize)
			{
				input2[j][idx] -= aver[idx];
			}
		}
	}
}

void 
	preProcessing(cuMatrixVector<float>&trainX, cuMatrixVector<float>&testX)
{
	int n_rows     = trainX[0]->rows;
	int n_cols     = trainX[0]->cols;
	int n_channels = trainX[0]->channels;

	cuMatrix<float>* aver(new cuMatrix<float>(n_rows, n_cols, n_channels));

	for (int imgId = 0; imgId < (int)trainX.size(); imgId++) {
		int len = trainX[0]->getLen();
		for (int i = 0; i < len; i++) {
			aver->getHost()[i] += trainX[imgId]->getHost()[i];
		}
	}

	for(int i = 0; i < aver->getLen(); i++){
		int len = trainX.size();
		aver->getHost()[i] /= len;
	}

	for (int imgId = 0; imgId < (int)trainX.size(); imgId++) {
		int len = trainX[0]->getLen();
		for (int i = 0; i < len; i++) {
			 trainX[imgId]->getHost()[i] -= aver->getHost()[i];
		}
	}

	aver->cpuClear();

	for (int imgId = 0; imgId < (int)testX.size(); imgId++) {
		int len = testX[0]->getLen();
		for (int i = 0; i < len; i++) {
			aver->getHost()[i] += testX[imgId]->getHost()[i];
		}
	}

	for(int i = 0; i < aver->getLen(); i++){
		int len = testX.size();
		aver->getHost()[i] /= len;
	}

	for (int imgId = 0; imgId < (int)testX.size(); imgId++) {
		int len = testX[0]->getLen();
		for (int i = 0; i < len; i++) {
			testX[imgId]->getHost()[i] -= aver->getHost()[i];
		}
	}

	delete aver;
}
